#include "hip/hip_runtime.h"
#include <array>
#include <iostream>
#include <limits>
#include <cassert>
#include <cstdio>

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

constexpr int BLOCK_SIZE = 256;

template<typename scalar_t, int dim>
using packed_accessor_t = torch::PackedTensorAccessor64<scalar_t, dim, torch::RestrictPtrTraits>;

// A table that stores precomputed powers `max_degree ^ x`
// It should be safe to initialize this with 64
// entries because the explicit form is unable to grow
// substantially when the number of variables is large.
constexpr int POW_TABLE_SIZE = 64;
__constant__ int64_t int_pow_table[POW_TABLE_SIZE];

__host__ __device__
int64_t int_pow(int base, int exp) {
  int64_t accum = 1;
  for (int i = 0; i < exp; ++i) {
    accum *= base;
  }
  return accum;
}

template<typename scalar_t>
__global__ 
void ibf_minmax_cuda_kernel(
	packed_accessor_t<scalar_t, 2> block_ebf_sum,
	const packed_accessor_t<scalar_t, 3> poly,
	const int nterms,
	const int nvars,
	const int max_degree,
	const int64_t ebf_chunk_start,
	const int64_t ebf_chunk_end) { 
  int term_id = blockIdx.x;
  while (term_id < nterms) {
    int ebf_id = ebf_chunk_start + threadIdx.x;
    while (ebf_id < ebf_chunk_end) {
      // ebf_id corresponds to a set of indices {i, j, k, ...}
      // ebf_id = i * d^{v-1} + j * d^{v-2} + k * d^{v-3} + ...
      // So, i = floor(ebf_id / d^{v-1})           (the lower order terms removed by floor)
      // Then, j = floor(ebf_id / d^{v-2}) - i * d
      // THen, k = floor(ebf_id / d^{v-3}) - i * d^2 - j * d
      scalar_t accum_prod = 1;
      int tracker = 0;
      for (int v = 0; v < nvars; ++v) {
	int64_t p = int_pow(max_degree, nvars - v - 1);
	//int64_t p = int_pow_table[nvars - v -1];
        int64_t index = (ebf_id / p) - tracker;
	accum_prod *= poly[term_id][v][index];
	tracker += index;
	tracker *= max_degree;
      } 
      block_ebf_sum[blockIdx.x][ebf_id - ebf_chunk_start] += accum_prod;
      ebf_id += blockDim.x;
    } 
    term_id += gridDim.x;
  }
}


/**
 * Compute the min/max of a Bernstein polynomial in implicit form.
 */ 
torch::Tensor ibf_minmax_cuda(torch::Tensor poly) {
  int nterms = poly.size(0);
  int nvars = poly.size(1);
  int max_degree = poly.size(2);
  int64_t ebf_size = int_pow(max_degree, nvars);

  // initialize int_pow_table
  std::vector<int64_t> max_degree_powers(POW_TABLE_SIZE);
  for (int i = 0; i < POW_TABLE_SIZE; ++i) {
    max_degree_powers[i] = int_pow(max_degree, i);
  }
  int64_t* pows = max_degree_powers.data();
  hipMemcpyToSymbol(HIP_SYMBOL(int_pow_table), pows, POW_TABLE_SIZE * sizeof(int64_t), 0, hipMemcpyHostToDevice);

  int64_t target_blocks_x = nterms;
  int64_t blocks_x = std::min(target_blocks_x, static_cast<int64_t>(int_pow(2, 30) / ebf_size));

  auto options = torch::TensorOptions()
    .dtype(poly.dtype())
    .device(poly.device());

  //int64_t ebf_chunk_size = int_pow(2, 6);
  //int64_t num_ebf_chunks = std::ceil(static_cast<float>(ebf_size) /
		 		     //static_cast<float>(ebf_chunk_size));

  //auto mins = torch::full(num_ebf_chunks, 99999, options);
  //auto maxs = torch::full(num_ebf_chunks, -99999, options);

  /*for (int ebf_chunk_start = 0; ebf_chunk_start < ebf_size; ebf_chunk_start += ebf_chunk_size) {
    int ebf_chunk_end = std::min(ebf_size, ebf_chunk_start + ebf_chunk_size);
    auto block_ebf_sum = torch::zeros({blocks_x, ebf_chunk_end - ebf_chunk_start}, options);
    const int blocks = blocks_x;
    const int threads = BLOCK_SIZE;
    AT_DISPATCH_FLOATING_TYPES(poly.type(), "ibf_cuda_minmax", ([&] {
      ibf_minmax_cuda_kernel<scalar_t><<<blocks, threads>>>(
        block_ebf_sum.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
        poly.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
        nterms,
        nvars,
        max_degree,
	ebf_chunk_start,
	ebf_chunk_end);
    }));
    */
  auto ebf_sum = torch::zeros({blocks_x, ebf_size}, options);
  const int threads = BLOCK_SIZE;
  
  AT_DISPATCH_FLOATING_TYPES(poly.type(), "ibf_cuda_minmax", ([&] {
    ibf_minmax_cuda_kernel<scalar_t><<<blocks_x, threads>>>(
      ebf_sum.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
      poly.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
      nterms,
      nvars,
      max_degree,
      0,
      ebf_size);
  }));

  auto sum = ebf_sum.sum(/*dim=*/0);
  auto minmax = torch::empty(2, options);
  minmax[0] = sum.min();
  minmax[1] = sum.max();

  return minmax;
}
