#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void repeatTerms2Kernel(float* TA, float* result, int tA, int n, int n_columns, int64_t rows_TA) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int64_t total_output_elements = n * (tA * tA + tA) / 2 * n_columns; // Total elements in the output tensor

    while (idx < total_output_elements) {
        int64_t col = idx % n_columns; // Column index
        int64_t output_row = idx / n_columns; // Row index in the output tensor

        // Determine the chunk and the position within the chunk
        int64_t chunk_index = 0, row_within_chunk = 0, accumulated_rows = 0;

        for (int i = 0; i < tA; i++) {
            int64_t rows_in_this_chunk = (tA - i) * n;
            if (output_row < accumulated_rows + rows_in_this_chunk) {
                chunk_index = i;
                row_within_chunk = output_row - accumulated_rows;
                break;
            }
            accumulated_rows += rows_in_this_chunk;
        }

        // Calculate the corresponding row in TA
        int64_t TA_row = chunk_index * n + row_within_chunk;
        if (TA_row < rows_TA) {
            result[idx] = TA[TA_row * n_columns + col];
            // Apply doubling condition
            if ((row_within_chunk >= n) && ((row_within_chunk - n) % n == 0)) {
                result[idx] *= 2;
            }
        } else {
            result[idx] = 0; // If out of bounds, set to zero
        }
	idx += gridDim.x;
    }
}

torch::Tensor repeat_terms_2_cuda(torch::Tensor TA, int tA, int n, int n_columns) {
    int64_t rows_TA = TA.size(0);
    int64_t total_output_rows = n * (tA * tA + tA) / 2;
    auto result = torch::empty({total_output_rows, n_columns}, TA.options());
    int threads = 1024;
    int blocks = static_cast<int>(std::min(static_cast<int64_t>(std::numeric_limits<int>::max()) - 1, 
		   	  		   static_cast<int64_t>(total_output_rows * n_columns + threads - 1) / threads));

    repeatTerms2Kernel<<<blocks, threads>>>(TA.data_ptr<float>(), result.data_ptr<float>(), tA, n, n_columns, rows_TA);

    return result;
}
