#include "hip/hip_runtime.h"
#include <array>
#include <iostream>
#include <limits>
#include <cassert>
#include <cstdio>

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

constexpr int BLOCK_SIZE_X = 32;
constexpr int BLOCK_SIZE_Y = 32;
constexpr int WARP_SIZE = 32;

template<typename scalar_t, int dim>
using packed_accessor_t = torch::PackedTensorAccessor64<scalar_t, dim, torch::RestrictPtrTraits>;

/**
 * This performs a reduction of data in a single GPU warp.
 * `op` can be any binary operation. 
 * The result of the reduction is stored in block[0]
 * This is based on these slides:
 * https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
 */
#define WARP_REDUCE(name,op)                                                          \
template<typename scalar_t, int block_size>                                           \
__device__                                                                            \
scalar_t name(volatile scalar_t* block, int local_tid) { 			      \
  if (block_size >= 64) block[local_tid] = op(block[local_tid], block[local_tid+32]); \
  if (block_size >= 32) block[local_tid] = op(block[local_tid], block[local_tid+16]); \
  if (block_size >= 16) block[local_tid] = op(block[local_tid], block[local_tid+8]);  \
  if (block_size >=  8) block[local_tid] = op(block[local_tid], block[local_tid+4]);  \
  if (block_size >=  4) block[local_tid] = op(block[local_tid], block[local_tid+2]);  \
  if (block_size >=  2) block[local_tid] = op(block[local_tid], block[local_tid+1]);  \
}

WARP_REDUCE(warp_min, std::min)
WARP_REDUCE(warp_max, std::max)

__host__ __device__
int64_t int_pow(int base, int exp) {
  int64_t accum = 1;
  for (int i = 0; i < exp; ++i) {
    accum *= base;
  }
  return accum;
}

template<typename scalar_t>
__global__ 
void ibf_minmax_cuda_kernel(
	packed_accessor_t<scalar_t, 2> block_ebf_sum,
	const packed_accessor_t<scalar_t, 3> poly,
	const int nterms,
	const int nvars,
	const int max_degree,
	const int64_t ebf_size) { 
  int tidx = threadIdx.x;
  int tidy = threadIdx.y;

  // global x idx determines the term.
  int term_id = blockIdx.x;

  while (term_id < nterms) {
    int ebf_id = threadIdx.x;
    while (ebf_id < ebf_size) {
      // ebf_id corresponds to a set of indices {i, j, k, ...}
      // ebf_id = i * d^{v-1} + j * d^{v-2} + k * d^{v-3} + ...
      // So, i = floor(ebf_id / d^{v-1})           (the lower order terms removed by floor)
      // Then, j = floor(ebf_id / d^{v-2}) - i * d
      // THen, k = floor(ebf_id / d^{v-3}) - i * d^2 - j * d
      scalar_t accum_prod = 1;
      int tracker = 0;
      for (int v = 0; v < nvars; ++v) {
        int64_t index = (ebf_id / int_pow(max_degree, nvars - v - 1)) - tracker;
	accum_prod *= poly[term_id][v][index];
	tracker += index;
	tracker *= max_degree;
      } 
      block_ebf_sum[blockIdx.x][ebf_id] += accum_prod;
      ebf_id += blockDim.x;
    } 
    term_id += gridDim.x;
  }

  //__syncthreads();

  //int col_id = threadIdx.x;
  //while (col_id < ebf_size) {
  //  col_id += blockDim.x;
  //}
}

/**
 * Compute the min/max of a Bernstein polynomial in implicit form.
 */ 
torch::Tensor ibf_minmax_cuda(torch::Tensor poly) {
  int nterms = poly.size(0);
  int nvars = poly.size(1);
  int max_degree = poly.size(2);
  int64_t ebf_size = int_pow(max_degree, nvars);

  int64_t target_blocks_x = nterms;
  int64_t blocks_x = std::min(target_blocks_x, static_cast<int64_t>(int_pow(2, 8)));

  auto options = torch::TensorOptions()
	  .dtype(poly.dtype())
	  .device(poly.device());

  auto block_ebf_sum = torch::zeros({blocks_x, ebf_size}, options);

  int blocks = blocks_x;
  int threads = BLOCK_SIZE_X;

  AT_DISPATCH_FLOATING_TYPES(poly.type(), "ibf_cuda_minmax", ([&] {
    ibf_minmax_cuda_kernel<scalar_t><<<blocks, threads>>>(
      block_ebf_sum.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
      poly.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
      nterms,
      nvars,
      max_degree,
      ebf_size);
  }));

  auto ebf_sum = block_ebf_sum.sum(/*dim=*/0);

  auto minmax = torch::empty(2, options);
  minmax[0] = std::get<0>(ebf_sum.min(0)).item();
  minmax[1] = std::get<0>(ebf_sum.max(0)).item();
  return minmax;
}
